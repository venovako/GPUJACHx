// main.cu: test driver.

#include "HypJac.hpp"
#include "HypJacL2.hpp"

#include "cuda_memory_helper.hpp"
#include "hdf5_helper.hpp"
#include "my_utils.hpp"

int main(int argc, char *argv[])
{
  if ((8 > argc) || (9 < argc)) {
    (void)fprintf(stderr, "%s DEV SDY SNP0 SNP1 ALG H5F H5G [H5R]\n", argv[0]);
    return EXIT_FAILURE;
  }

  const char *const ca_exe = argv[0];
  const char *const ca_dev = argv[1];
  const char *const ca_sdy = argv[2];
  const char *const ca_snp0 = argv[3];
  const char *const ca_snp1 = argv[4];
  const char *const ca_alg = argv[5];
  const char *const ca_h5f = argv[6];
  const char *const ca_h5g = argv[7];
  const char *const ca_h5r = ((9 == argc) ? argv[8] : static_cast<const char*>(NULL));

  unsigned idadim[HYPJAC_IDADIM_SIZE] = { 0u };
  hid_t fid = static_cast<hid_t>(-1), gid = static_cast<hid_t>(-1);
  size_t ldA = static_cast<size_t>(0u), m = static_cast<size_t>(0u), n = static_cast<size_t>(0u);

  const unsigned n0 = (HYPJACL1_NCOLB << 1u);
  const unsigned n1 = (static_cast<unsigned>(atoi(ca_h5g)) + HYPJACL1_NCOLB - 1u) / HYPJACL1_NCOLB;
  init_strats(ca_sdy, ca_snp0, n0, ca_snp1, n1);

  const int dev = atoi(ca_dev);
  const int dcc = configureGPU(dev);
  if (dcc < 30) {
    (void)snprintf(err_msg, err_msg_size, "Device %d has CC %d < 30", dev, dcc);
    DIE(err_msg);
  }

  const unsigned routine = static_cast<unsigned>(atoi(ca_alg));

  if (!fexist(ca_h5f)) {
    (void)snprintf(err_msg, err_msg_size, "non-existent H5F(%s)", ca_h5f);
    DIE(err_msg);
  }

  HDF5_CALL(H5open());

  HDF5_CALL((fid = H5Fopen(ca_h5f, H5F_ACC_RDONLY, H5P_DEFAULT)));
  HDF5_CALL((gid = H5Gopen2(fid, ca_h5g, H5P_DEFAULT)));

  HDF5_CALL(H5LTread_dataset_int(gid, "IDADIM", reinterpret_cast<int*>(idadim)));

  unsigned
    ldhG = idadim[0],
    nrow = idadim[1],
    ncol = idadim[2],
    nplus = idadim[3];

  m = static_cast<size_t>(nrow);
  n = static_cast<size_t>(ncol);

  ldA = static_cast<size_t>(ldhG);
  double *const hG = allocHostMtx<double>(ldA, m, n, true);
  SYSP_CALL(hG);
  ldhG = static_cast<unsigned>(ldA);

  HDF5_CALL(H5LTread_dataset_double(gid, "G", hG));

  HDF5_CALL(H5Gclose(gid));
  HDF5_CALL(H5Fclose(fid));

  double *hV;
  unsigned ldhV;
  if (routine & HYPJAC_FULL_SVD) {
    hV = allocHostMtx<double>(ldA, n, n, true);
    SYSP_CALL(hV);
    ldhV = static_cast<unsigned>(ldA);
  }
  else {
    hV = static_cast<double*>(NULL);
    ldhV = 0u;
  }

  double *const hD = allocHostVec<double>(n);
  SYSP_CALL(hD);

  unsigned glbSwp = 0u;
  unsigned long long glb_s = 0ull, glb_b = 0ull;
  double timing[4] = { -0.0, -0.0, -0.0, -0.0 };
  int ret = hypJacL2(routine, nrow, ncol, nplus, hG, ldhG, hV, ldhV, hD, &glbSwp, &glb_s, &glb_b, timing);

  if (ret)
    (void)fprintf(stderr, "%s: error %d\n", ca_exe, ret);
  else {
    (void)fprintf(stdout, "GLB_ROT_S(%20llu), GLB_ROT_B(%20llu)\n", glb_s, glb_b);
    (void)fflush(stdout);
    (void)fprintf(stdout, "%#12.6f s %2u sweeps\n", *timing, glbSwp);
    (void)fflush(stdout);
  }

  if (ca_h5r) {
    HDF5_CALL(fid = fexist(ca_h5r) ?
              H5Fopen(ca_h5r, H5F_ACC_RDWR, H5P_DEFAULT) :
              H5Fcreate(ca_h5r, H5F_ACC_EXCL, H5P_DEFAULT, H5P_DEFAULT));
    HDF5_CALL(gid = H5Gcreate2(fid, ca_h5g, H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT));

    idadim[0] = ldhG;
    hsize_t dims[2] = { static_cast<hsize_t>(HYPJAC_IDADIM_SIZE), static_cast<hsize_t>(1u) };
    HDF5_CALL(H5LTmake_dataset_int(gid, "IDADIM", 1, dims, reinterpret_cast<int*>(idadim)));

    int istats[4] = { static_cast<int>(routine), static_cast<int>(STRAT0), static_cast<int>(STRAT1), static_cast<int>(ret) };
    istats[0] = ((ret < 0) ? ret : static_cast<int>(glbSwp));
    dims[0] = static_cast<hsize_t>(4u);
    dims[1] = static_cast<hsize_t>(1u);
    HDF5_CALL(H5LTmake_dataset_int(gid, "ISTATS", 1, dims, istats));

    dims[0] = static_cast<hsize_t>(4u);
    dims[1] = static_cast<hsize_t>(1u);
    HDF5_CALL(H5LTmake_dataset_double(gid, "TIMING", 1, dims, timing));
    
    if (hG) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhG);
      HDF5_CALL(H5LTmake_dataset_double(gid, "G", 2, dims, hG));
    }

    if (hV) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhV);
      HDF5_CALL(H5LTmake_dataset_double(gid, "V", 2, dims, hV));
    }

    if (hD) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "D", 1, dims, hD));
    }

    HDF5_CALL(H5Gclose(gid));
    HDF5_CALL(H5Fclose(fid));
  }

  HDF5_CALL(H5close());

  if (hD)
    CUDA_CALL(hipHostFree(hD));
  if (hV)
    CUDA_CALL(hipHostFree(hV));
  if (hG)
    CUDA_CALL(hipHostFree(hG));

  // for profiling
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipDeviceReset());

  return ret;
}
