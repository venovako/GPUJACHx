#include "cuda_helper.hpp"

#include "my_utils.hpp"

int configureGPUex(const int dev, const unsigned maxShMemB) throw()
{
  assert(dev >= 0);

  CUDA_CALL(hipSetDeviceFlags(hipDeviceMapHost | hipDeviceScheduleSpin));
  CUDA_CALL(hipSetDevice(dev));

  hipDeviceProp_t cdp;
  CUDA_CALL(hipGetDeviceProperties(&cdp, dev));
  const int dcc = cdp.major * 10 + cdp.minor;

  if (dcc < 20) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d Compute Capability %d.%d < 2.0", dev, cdp.major, cdp.minor);
    DIE(err_msg);
  }

  if (WARP_SZ != static_cast<unsigned>(cdp.warpSize)) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d has %d threads in a warp, must be %u", dev, cdp.warpSize, WARP_SZ);
    DIE(err_msg);
  }

  if (!cdp.unifiedAddressing) {
    (void)snprintf(err_msg, err_msg_size, "CUDA Device %d does not support unified addressing", dev);
    DIE(err_msg);
  }

  hipFuncCache_t cacheConfig = hipFuncCachePreferNone;
  if (maxShMemB <= 16384u) // 16 kB
    cacheConfig = hipFuncCachePreferL1;
  else if (maxShMemB <= 32768u) // 32 kB
    cacheConfig = hipFuncCachePreferEqual;
  else if (maxShMemB <= 49152u) // 48 kB
    cacheConfig = hipFuncCachePreferShared;
  else { // > 48 kB
    (void)snprintf(err_msg, err_msg_size, "Maximum shared memory requested (%u B) > 48 kB", maxShMemB);
    WARN(err_msg);
  }
  CUDA_CALL(hipDeviceSetCacheConfig(cacheConfig));

  if (dcc >= 30)
    CUDA_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte));

  return dcc;
}

int configureGPU(const int dev) throw()
{
  static const unsigned maxShMemB =
#ifdef HZ_GSVD
    24576u // 24 kB
#else // Jacobi (H)SVD
    16384u // 16 kB
#endif // HZ_GSVD
    ;
  return configureGPUex(dev, maxShMemB);
}
