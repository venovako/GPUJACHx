// main.cu: test driver.

#include "HZ.hpp"
#include "HZ_L2.hpp"

#include "cuda_memory_helper.hpp"
#include "hdf5_helper.hpp"
#include "my_utils.hpp"

struct CmdArgs {
  char *exe;
  char *dev;
  char *sdy;
  char *snp;
  char *alg;
  char *h5f;
  char *h5g;
  char *h5r;

  CmdArgs(const int argc, char *const argv[]) throw()
    : exe(static_cast<char*>(NULL)),
      dev(static_cast<char*>(NULL)),
      sdy(static_cast<char*>(NULL)),
      snp(static_cast<char*>(NULL)),
      alg(static_cast<char*>(NULL)),
      h5f(static_cast<char*>(NULL)),
      h5g(static_cast<char*>(NULL)),
      h5r(static_cast<char*>(NULL))
  {
    if (1 > argc)
      DIE("argc <= 0");
    if (!argv)
      DIE("NULL argv");

    if ((7 > argc) || (8 < argc)) {
      (void)fprintf(stderr, "%s DEV SDY SNP ALG H5F H5G [H5R]\n", argv[0]);
      exit(EXIT_FAILURE);
    }

    exe = argv[0];
    dev = argv[1];
    sdy = argv[2];
    snp = argv[3];
    alg = argv[4];
    h5f = argv[5];
    h5g = argv[6];
    h5r = ((8 == argc) ? argv[7] : static_cast<char*>(NULL));
  }
};

int main(int argc, char *argv[])
{
  int ret = EXIT_SUCCESS;

  unsigned idadim[HZ_IDADIM_SIZE] = { 0u };
  hid_t fid = static_cast<hid_t>(-1), gid = static_cast<hid_t>(-1);
  size_t ldA = static_cast<size_t>(0u), m = static_cast<size_t>(0u), n = static_cast<size_t>(0u);

  CmdArgs ca(argc, argv);

  const unsigned n0 = (HZ_L1_NCOLB << 1u);
  const unsigned n1 = (static_cast<unsigned>(atoi(ca.h5g)) + HZ_L1_NCOLB - 1u) / HZ_L1_NCOLB;
  init_strats(ca.sdy, ca.snp, n0, n1);

  const int dev = atoi(ca.dev);
  const int dcc = configureGPU(dev);
  if (dcc < 30) {
    (void)snprintf(err_msg, err_msg_size, "Device %d has CC %d < 30", dev, dcc);
    DIE(err_msg);
  }

  const unsigned routine = static_cast<unsigned>(atoi(ca.alg));

  if (!fexist(ca.h5f)) {
    (void)snprintf(err_msg, err_msg_size, "non-existent H5F(%s)", ca.h5f);
    DIE(err_msg);
  }

  HDF5_CALL(H5open());

  HDF5_CALL((fid = H5Fopen(ca.h5f, H5F_ACC_RDONLY, H5P_DEFAULT)));
  HDF5_CALL((gid = H5Gopen2(fid, ca.h5g, H5P_DEFAULT)));

  HDF5_CALL(H5LTread_dataset_int(gid, "IDADIM", reinterpret_cast<int*>(idadim)));

  unsigned
    ldhF = idadim[0],
    ldhG = idadim[0],
    ldhV = 0u,
    nrow = idadim[0], //idadim[1]
    ncol = idadim[0]; //idadim[2]

  m = static_cast<size_t>(nrow);
  n = static_cast<size_t>(ncol);

  ldA = static_cast<size_t>(ldhF);
  double *const hF = allocHostMtx<double>(ldA, m, n, true);
  SYSP_CALL(hF);
  ldhF = static_cast<unsigned>(ldA);

  HDF5_CALL(H5LTread_dataset_double(gid, "F", hF));

  ldA = static_cast<size_t>(ldhG);
  double *const hG = allocHostMtx<double>(ldA, m, n, true);
  SYSP_CALL(hG);
  ldhG = static_cast<unsigned>(ldA);

  HDF5_CALL(H5LTread_dataset_double(gid, "G", hG));

  HDF5_CALL(H5Gclose(gid));
  HDF5_CALL(H5Fclose(fid));

  double *hV = static_cast<double*>(NULL);
  ldhV = ((ldhF <= ldhG) ? ldhF : ldhG);
  ldA = static_cast<size_t>(ldhV);
  hV = allocHostMtx<double>(ldA, n, n, true);
  SYSP_CALL(hV);
  ldhV = static_cast<unsigned>(ldA);

  double *const hS = allocHostVec<double>(n);
  SYSP_CALL(hS);
  double *const hH = allocHostVec<double>(n);
  SYSP_CALL(hH);
  double *const hK = allocHostVec<double>(n);
  SYSP_CALL(hK);

  unsigned glbSwp = 0u;
  unsigned Long glb_s = MkLong(0u), glb_b = MkLong(0u);
  double timing[4] = { -0.0, -0.0, -0.0, -0.0 };
  ret = HZ_L2(routine, nrow, ncol, hF, ldhF, hG, ldhG, hV, ldhV, hS, hH, hK, &glbSwp, &glb_s, &glb_b, timing);

  if (ret)
    (void)fprintf(stderr, "%s: error %d\n", ca.exe, ret);
  else {
    (void)printf("GLB_ROT_S(%20" FmtLong "u), GLB_ROT_B(%20" FmtLong "u)\n", glb_s, glb_b);
    (void)printf("%#12.6f s %2u sweeps\n", *timing, glbSwp);
  }

  if (ca.h5r) {
    HDF5_CALL(fid = fexist(ca.h5r) ?
              H5Fopen(ca.h5r, H5F_ACC_RDWR, H5P_DEFAULT) :
              H5Fcreate(ca.h5r, H5F_ACC_EXCL, H5P_DEFAULT, H5P_DEFAULT));
    HDF5_CALL(gid = H5Gcreate2(fid, ca.h5g, H5P_DEFAULT, H5P_DEFAULT, H5P_DEFAULT));

    idadim[0] = ldhG;
    hsize_t dims[2] = { static_cast<hsize_t>(HZ_IDADIM_SIZE), static_cast<hsize_t>(1u) };
    HDF5_CALL(H5LTmake_dataset_int(gid, "IDADIM", 1, dims, reinterpret_cast<int*>(idadim)));

    int istats[4] = { 0, static_cast<int>(STRAT0), static_cast<int>(STRAT1), dev };
    istats[0] = ((ret < 0) ? ret : static_cast<int>(glbSwp));
    dims[0] = static_cast<hsize_t>(4u);
    dims[1] = static_cast<hsize_t>(1u);
    HDF5_CALL(H5LTmake_dataset_int(gid, "ISTATS", 1, dims, istats));

    dims[0] = static_cast<hsize_t>(4u);
    dims[1] = static_cast<hsize_t>(1u);
    HDF5_CALL(H5LTmake_dataset_double(gid, "TIMING", 1, dims, timing));

    if (hF) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhF);
      HDF5_CALL(H5LTmake_dataset_double(gid, "F", 2, dims, hF));
    }

    if (hG) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhG);
      HDF5_CALL(H5LTmake_dataset_double(gid, "G", 2, dims, hG));
    }

    if (hV) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(ldhV);
      HDF5_CALL(H5LTmake_dataset_double(gid, "V", 2, dims, hV));
    }

    if (hS) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "SIGMA", 1, dims, hS));
    }

    if (hH) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "H", 1, dims, hH));
    }

    if (hK) {
      dims[0] = static_cast<hsize_t>(ncol);
      dims[1] = static_cast<hsize_t>(1u);
      HDF5_CALL(H5LTmake_dataset_double(gid, "K", 1, dims, hK));
    }

    HDF5_CALL(H5Gclose(gid));
    HDF5_CALL(H5Fclose(fid));
  }

  HDF5_CALL(H5close());

  if (hK)
    CUDA_CALL(hipHostFree(hK));
  if (hH)
    CUDA_CALL(hipHostFree(hH));
  if (hS)
    CUDA_CALL(hipHostFree(hS));
  if (hV)
    CUDA_CALL(hipHostFree(hV));
  if (hG)
    CUDA_CALL(hipHostFree(hG));
  if (hF)
    CUDA_CALL(hipHostFree(hF));

  // for profiling
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipDeviceReset());

  return ret;
}
